
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime_api.h>
#define funcCheck(stmt) do {                                                  \
        hipError_t err = stmt;                                               \
        if (err != hipSuccess) {                                             \
            printf( "Failed to run stmt %d ", __LINE__);                      \
            printf( "Got CUDA error ...  %s ", hipGetErrorString(err));      \
            return -1;                                                        \
        }                                                                     \
    } while(0)


__global__ void convolution_1D(float *N,float *M,float *P,int Mask_width,int width)
{
int i=blockIdx.x*blockDim.x+threadIdx.x;
float Pvalue=0.0;
int N_start_point=i-(Mask_width/2);
for(int j=0;j<Mask_width;j++)
{
 if(((N_start_point+j)>=0) && ((N_start_point+j)<width))
 {
  Pvalue+=N[N_start_point+j]*M[j];
 }
}
P[i]=Pvalue;
}


int main()
{
 float * input;
 float * Mask;
 float * output;
 float * device_input;
 float * device_Mask;
 float * device_output;
 int Mask_width=3;
 int width=5;

 input=(float *)malloc(sizeof(float)*width);
 Mask=(float *)malloc(sizeof(float)*Mask_width);
 output=(float *)malloc(sizeof(float)*width);
 for(int i=0;i<width;i++)
 {
  input[i]=1.0;
 }
 for(int i=0;i<Mask_width;i++)
 {
  Mask[i]=1.0;
 }

 printf("\nInput:\n");
 for(int i=0;i<width;i++)
 {
   printf("%0.2f ",input[i]);
 }printf("\n");

 printf("\nMask:\n");

  for(int i=0;i<Mask_width;i++)
  {
    printf("%0.2f ",Mask[i]);
  }printf("\n");

  funcCheck(hipMalloc((void **)&device_input,sizeof(float)*width));
  funcCheck(hipMalloc((void **)&device_Mask,sizeof(float)*Mask_width));
  funcCheck(hipMalloc((void **)&device_output,sizeof(float)*width));

  funcCheck(hipMemcpy(device_input,input,sizeof(float)*width,hipMemcpyHostToDevice));
  funcCheck(hipMemcpy(device_Mask,Mask,sizeof(float)*Mask_width,hipMemcpyHostToDevice));

  dim3 dimGrid(((width-1)/Mask_width)+1, 1,1);
  dim3 dimBlock(Mask_width,1, 1);

  convolution_1D<<<dimGrid,dimBlock>>>(device_input,device_Mask,device_output,Mask_width,width);

  hipError_t err1 = hipPeekAtLastError();
  hipDeviceSynchronize();

  printf( "Got CUDA error ... %s \n", hipGetErrorString(err1));
  funcCheck(hipMemcpy(output,device_output,sizeof(float)*width,hipMemcpyDeviceToHost));


 printf("\n\nOutput: \n");

 for(int i=0;i<width;i++)
  {
   printf(" %0.2f \t",*(output+i));
  }

 hipFree(device_input);
 hipFree(device_Mask);
 hipFree(device_output);

 free(input);
 free(output);
 free(Mask);


 printf("\n \nNumber of Blocks Created :%d",(((width-1)/Mask_width)+1));
 printf("\n \nNumber of Threads Per Block created in code: %d",(Mask_width));

 return 0;
}

 
